#include "hip/hip_runtime.h"
/**
 * @file    main.cu
 * @author  Yohai Meiron <ymeiron@pku.edu.cn>
 * @version 1.0
 */

// If you have CUDA then compile with:
// nvcc main.cu -lm -O3 -arch=sm_20 -o output
// Otherwise enable OpenMP and compile with GCC:
// g++ -x c++ -O3 -o output main.cu -DOMP -fopenmp -lgomp -I/home/ym
// The -I is the path to the parent directory where thrust is.

#include <iostream>
#include <fstream>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <mpi.h>

#define CUDA

#ifdef OMP
    #error Sorry, OpenMP is currently disabled.
    #define THRUST_DEVICE_SYSTEM THRUST_DEVICE_BACKEND_OMP
    #undef CUDA
    #define PARALLEL_GET_TID omp_get_thread_num()
    #define PARALLEL_ADVANCE omp_get_num_threads()
    #define __global__
//     #include <complex>
#else
    #define PARALLEL_GET_TID threadIdx.x + blockIdx.x * blockDim.x
    #define PARALLEL_ADVANCE blockDim.x * gridDim.x
//     #include "cuda_complex.hpp"
#endif
// #define complex complex<Real>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/partition.h>
#include <thrust/inner_product.h>

#include "common.hpp"
#include "io.hpp"
#include "ic.hpp"
#include "integrate.hpp"

#ifdef MEX
    #define method mex
    #include "mex.hpp"
#elif defined(SCF)
    #define method scf
    #include "scf.hpp"
#endif

using namespace std;
using namespace etics;

// GLOBAL VARIABLES
int MyRank, NumProcs;
Real ConstantStep = 0.001953125;
Real T, Step, dT1, dT2, Tcrit, FileTime;
int NSteps = 0, FileSnapshotNum;

struct ReorderingFunctor {
    __host__ __device__ bool operator() (const Particle &lhs, const Particle &rhs) {
        return (lhs.ID <= rhs.ID);
    }
};

Real CalculateStepSize() {
    return ConstantStep;
}

void DisplayInformation(Integrator IntegratorObj) {
    Real Ek = IntegratorObj.KineticEnergy();
    Real Ep = IntegratorObj.PotentialEnergy();
    Real Energy = Ek + Ep;

    Real TotalEnergy;
    MPI_Reduce(&Energy, &TotalEnergy, 1, MPI_ETICS_REAL, MPI_SUM, 0, MPI_COMM_WORLD);

    int N=IntegratorObj.GetN(), TotalN;
    MPI_Reduce(&N, &TotalN, 1, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);

    if (MyRank==0) {
        printf(" TIME =%6.2f  NSTEPS =%6d  ENERGY =%20.16f   N = %d\n", T, NSteps, TotalEnergy, TotalN);
        fflush(stdout);
    }
}

void PrepareSnapshot(Integrator IntegratorObj, Particle **ParticleList, int *CurrentTotalN) {
    Particle *LocalList;
    int LocalBufferSize;
    IntegratorObj.CopyParticlesToHost(&LocalList, &LocalBufferSize);
    LocalBufferSize *= sizeof(Particle);
    int BufferSizes[NumProcs];
    MPI_Gather(&LocalBufferSize, 1, MPI_INT, BufferSizes, 1, MPI_INT, 0, MPI_COMM_WORLD);
    int Displacements[NumProcs];
    int TotalN = 0;
    if (MyRank==0) {
        for (int p = 0; p < NumProcs; p++) TotalN += BufferSizes[p]/sizeof(Particle);
        Displacements[0] = 0;
        for (int p = 1; p < NumProcs; p++) Displacements[p] = Displacements[p-1] + BufferSizes[p-1];
        *ParticleList = new Particle[TotalN];
    }
    MPI_Gatherv(LocalList, LocalBufferSize, MPI_BYTE, *ParticleList, BufferSizes,
 Displacements, MPI_BYTE, 0, MPI_COMM_WORLD);
#ifdef MEX
    thrust::sort(*ParticleList, (*ParticleList)+TotalN, ReorderingFunctor());
#endif
    *CurrentTotalN = TotalN;
}

int main(int argc, char *argv[]) {
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &MyRank);
    MPI_Comm_size(MPI_COMM_WORLD, &NumProcs);

    if (MyRank==0) {
        cerr << "Welcome to ETICS..." << endl;
#ifdef MEX
        cerr << "Using method: MEX" << endl;
        cerr << "LMAX=" << LMAX << endl;
#elif defined(SCF)
        cerr << "Using method: SCF" << endl;
        cerr << "LMAX=" << LMAX << endl;
        cerr << "NMAX=" << NMAX << endl;
#endif
    }

    string Filename;
    int DeviceID = 0;

    ParametersStruct Params;
    // Instead of reading the input file with MyRank=0 and broadcast the result, we let every rank read the file. This probably saves ~20 lines of ugly MPI code.
    ParseInput(argc, argv, &Params);
    int N = Params.N; // total; will be divided by number of processes
    Filename = Params.Filename;
    Tcrit = Params.Tcrit;
    ConstantStep = Params.ConstantStep;
    DeviceID = Params.DeviceID;
    dT1 = Params.dT1;
    dT2 = Params.dT2;

    if (DeviceID >= 0) {
        if (hipSetDevice(DeviceID) != hipSuccess) {
            cerr <<  "Problem opening device (ID=" << DeviceID << ")" << endl;
            exit(1);
        }
    } else {
        cerr << "Skipping call to hipSetDevice." << endl;
    }

    // Read an input file and initialize the global particle structure.
    Particle *FullList;
    if (MyRank==0) {
        if ((Filename == "_nofile_") || (Filename == "_hernquist_")) {
            cout << "Generating a Hernquist sphere..." << endl;
            etics::ic::hernquist(N, Params.Seed, &FullList);
            FileSnapshotNum = 0;
            FileTime = 0;
            cout << "Done." << endl;
        } else if (Filename == "_plummer_") {
            cout << "Generating a Plummer sphere..." << endl;
            etics::ic::plummer(N, Params.Seed, &FullList);
            FileSnapshotNum = 0;
            FileTime = 0;
            cout << "Done." << endl;
        } else if (Filename == "_launch_config_") {
            if (NumProcs > 1) {
                cerr << "Can only optimize launch configuration when a single MPI process is running!" << endl;
                exit(1);
            }
            int k3gs, k3bs, k4gs, k4bs;
            scf::OptimizeLaunchConfiguration(N, &k3gs, &k3bs, &k4gs, &k4bs);
            return 0;
        }
        else {
            string InputFileSuffix = Filename.substr(Filename.find_last_of("."), Filename.length()-Filename.find_last_of("."));

            if ((InputFileSuffix==".h5part") || (InputFileSuffix==".hdf5") || (InputFileSuffix==".h5")) {
#ifndef ETICS_HDF5
                cerr << "Compiled without the \"ETICS_HDF5\" flag; cannot read input in this format." << endl;
                exit(1);
#else
                ReadICsHDF5(Filename, N, &FullList, &FileSnapshotNum, &FileTime);
#endif
            } else ReadICsASCII(Filename, N, &FullList, &FileSnapshotNum, &FileTime);
        }
    }
    
#ifndef ETICS_HDF5
    if (Params.Format == "hdf5") {
        cerr << "Compiled without the \"ETICS_HDF5\" flag; cannot output in requested format." << endl;
        exit(1);
    }
#endif
    if (!(Params.Format == "hdf5") && !(Params.Format == "ascii")) {
        cerr << "Requested output format unrecognized." << endl;
        exit(1);
    }

    int LocalN = N / NumProcs;
    int Remainder = N - LocalN*NumProcs;
    if (MyRank==NumProcs-1) LocalN += Remainder;
    Particle *LocalList = new Particle[LocalN];
    int BufferSizes[NumProcs];
    int Displacements[NumProcs];
    if (MyRank==0) {
        for (int p = 0; p < NumProcs; p++) BufferSizes[p] = (N / NumProcs)*sizeof(Particle);
        BufferSizes[NumProcs-1] += Remainder*sizeof(Particle);
        Displacements[0] = 0;
        for (int p = 1; p < NumProcs; p++) Displacements[p] = Displacements[p-1] + BufferSizes[p-1];
    }
    MPI_Scatterv(FullList, BufferSizes, Displacements, MPI_BYTE, LocalList, LocalN*sizeof(Particle), MPI_BYTE, 0, MPI_COMM_WORLD);

    if (MyRank==0) free(FullList);
    N = LocalN;

    method::Init(N, 180, 64, 2605, 384);
#warning hardcoded launch configuration
    Integrator IntegratorObj(LocalList, N);

    // More initializations.
    Real NextOutput = 0, NextSnapshot = 0;
    T = FileTime;
    int SnapNumber = FileSnapshotNum;
    Step = CalculateStepSize();

    while (T <= Tcrit) {
        if (T >= NextOutput) {
            DisplayInformation(IntegratorObj);
            NextOutput += dT1;
        }
        if (T >= NextSnapshot) {
            int CurrentTotalN;
            PrepareSnapshot(IntegratorObj, &FullList, &CurrentTotalN);
            if (MyRank==0) {
                WriteSnapshotASCII(Params.Prefix, SnapNumber, FullList, CurrentTotalN, T);
                free(FullList);
            }
            SnapNumber++;
            NextSnapshot += dT2;
        }

        // Take the drift step.
        IntegratorObj.DriftStep(Step);

        // Calculate the forces in the new positions.
        IntegratorObj.CalculateGravity();

        // Finish by taking the kick step.
        // The kick functor also "commits" the predicted forces into the "acc" member.
        IntegratorObj.KickStep(Step);

        // N particles were implicitly propagated in this iteration.
        NSteps += 1;

        // Advance global time.
        T += Step;

        // Calculate the next step.
        Step = CalculateStepSize();
    }
    IntegratorObj.~Integrator();
    MPI_Finalize();
    return 0;
}
